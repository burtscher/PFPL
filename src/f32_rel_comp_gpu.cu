#include "hip/hip_runtime.h"
/*
This file is part of the LC framework for synthesizing high-speed parallel lossless and error-bounded lossy data compression and decompression algorithms for CPUs and GPUs.

BSD 3-Clause License

Copyright (c) 2021-2025, Noushin Azami, Alex Fallin, Brandon Burtchell, Andrew Rodriguez, Benila Jerald, Yiqian Liu, and Martin Burtscher
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

URL: The latest version of this code is available at https://github.com/burtscher/LC-framework.

Sponsor: This code is based upon work supported by the U.S. Department of Energy, Office of Science, Office of Advanced Scientific Research (ASCR), under contract DE-SC0022223.
*/


#define NDEBUG

using byte = unsigned char;
static const int CS = 1024 * 16;  // chunk size (in bytes) [must be multiple of 8]
static const int TPB = 512;  // threads per block [must be power of 2 and at least 128]
#define WS 32


#include <string>
#include <cmath>
#include <cassert>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include "include/macros.h"
#include "include/sum_reduction.h"
#include "include/max_scan.h"
#include "include/prefix_sum.h"
#include "components/d_DIFFNB_4.h"
#include "components/d_BIT_4.h"
#include "components/d_RZE_1.h"
#include <sys/time.h>


const int NUM_RUNS = 9;


static __device__ inline float d_REL_log2approxf(const float orig_f)
{
  //assert(orig_f > 0);
  const int mantissabits = 23;
  const int orig_i = *((int*)&orig_f);
  const int expo = (orig_i >> mantissabits) & 0xff;
  //if ((expo == 0) || (expo == 0xff)) return orig_f;
  const int frac_i = (127 << mantissabits) | (orig_i & ~(~0 << mantissabits));
  const float frac_f = *((float*)&frac_i);
  const float log_f = frac_f + (expo - 128);  // - bias - 1
  return log_f;
}


static __device__ inline float d_REL_pow2approxf(const float log_f)
{
  const int mantissabits = 23;
  const float biased = log_f + 127;
  const int expo = biased;
  const float frac_f = biased - (expo - 1);
  const int frac_i = *((int*)&frac_f);
  const int exp_i = (expo << mantissabits) | (frac_i & ~(~0 << mantissabits));
  const float recon_f = *((float*)&exp_i);
  return recon_f;
}


// copy (len) bytes from shared memory (source) to global memory (destination)
// source must we word aligned
static inline __device__ void s2g(void* const __restrict__ destination, const void* const __restrict__ source, const int len)
{
  const int tid = threadIdx.x;
  const byte* const __restrict__ input = (byte*)source;
  byte* const __restrict__ output = (byte*)destination;
  if (len < 128) {
    if (tid < len) output[tid] = input[tid];
  } else {
    const int nonaligned = (int)(size_t)output;
    const int wordaligned = (nonaligned + 3) & ~3;
    const int linealigned = (nonaligned + 127) & ~127;
    const int bcnt = wordaligned - nonaligned;
    const int wcnt = (linealigned - wordaligned) / 4;
    const int* const __restrict__ in_w = (int*)input;
    if (bcnt == 0) {
      int* const __restrict__ out_w = (int*)output;
      if (tid < wcnt) out_w[tid] = in_w[tid];
      for (int i = tid + wcnt; i < len / 4; i += TPB) {
        out_w[i] = in_w[i];
      }
      if (tid < (len & 3)) {
        const int i = len - 1 - tid;
        output[i] = input[i];
      }
    } else {
      const int shift = bcnt * 8;
      const int rlen = len - bcnt;
      int* const __restrict__ out_w = (int*)&output[bcnt];
      if (tid < bcnt) output[tid] = input[tid];
      if (tid < wcnt) out_w[tid] = __funnelshift_r(in_w[tid], in_w[tid + 1], shift);
      for (int i = tid + wcnt; i < rlen / 4; i += TPB) {
        out_w[i] = __funnelshift_r(in_w[i], in_w[i + 1], shift);
      }
      if (tid < (rlen & 3)) {
        const int i = len - 1 - tid;
        output[i] = input[i];
      }
    }
  }
}


static __device__ int g_chunk_counter;


static __global__ void d_reset()
{
  g_chunk_counter = 0;
}


static inline __device__ void propagate_carry(const int value, const int chunkID, volatile int* const __restrict__ fullcarry, int* const __restrict__ s_fullc)
{
  if (threadIdx.x == TPB - 1) {  // last thread
    fullcarry[chunkID] = (chunkID == 0) ? value : -value;
  }

  if (chunkID != 0) {
    if (threadIdx.x + WS >= TPB) {  // last warp
      const int lane = threadIdx.x % WS;
      const int cidm1ml = chunkID - 1 - lane;
      int val = -1;
      __syncwarp();  // not optional
      do {
        if (cidm1ml >= 0) {
          val = fullcarry[cidm1ml];
        }
      } while ((__any_sync(~0, val == 0)) || (__all_sync(~0, val <= 0)));
#if defined(WS) && (WS == 64)
      const long long mask = __ballot_sync(~0, val > 0);
      const int pos = __ffsll(mask) - 1;
#else
      const int mask = __ballot_sync(~0, val > 0);
      const int pos = __ffs(mask) - 1;
#endif
      int partc = (lane < pos) ? -val : 0;
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800)
      partc = __reduce_add_sync(~0, partc);
#else
      partc += __shfl_xor_sync(~0, partc, 1);
      partc += __shfl_xor_sync(~0, partc, 2);
      partc += __shfl_xor_sync(~0, partc, 4);
      partc += __shfl_xor_sync(~0, partc, 8);
      partc += __shfl_xor_sync(~0, partc, 16);
#endif
      if (lane == pos) {
        const int fullc = partc + val;
        fullcarry[chunkID] = fullc + value;
        *s_fullc = fullc;
      }
    }
  }
}


static __device__ inline void d_QREL_4(int& csize, byte in [CS], byte out [CS], const float errorbound, const float threshold)
{
  using ftype = float;
  using itype = int;
  const int size = csize / sizeof(ftype);
  const int tid = threadIdx.x;

  const int mantissabits = 23;
  const itype signexpomask = ~0 << mantissabits;
  const itype maxbin = (1 << (mantissabits - 2)) - 1;  // leave 2 bits for 2 signs (plus one element)

  const ftype log2eb = 2 * d_REL_log2approxf(1 + errorbound);
  const ftype inv_log2eb = 1 / log2eb;

  itype* const data_out_i = (itype*)out;
  itype* const data_in_i = (itype*)in;

  for (int i = tid; i < size; i += TPB) {
    const itype orig_i = data_in_i[i];
    const itype abs_orig_i = orig_i & 0x7fff'ffff;
    const ftype abs_orig_f = *((ftype*)&abs_orig_i);
    itype output = orig_i;
    const int expo = (orig_i >> mantissabits) & 0xff;
    if (expo == 0) {  // zero or de-normal values
      if (abs_orig_i == 0) {  // zero
        output = signexpomask | 1;
      }
    } else {
      if (expo == 0xff) {  // INF or NaN
        if (((orig_i & signexpomask) == signexpomask) && ((orig_i & ~signexpomask) != 0)) {  // negative NaN
          output = abs_orig_i;  // make positive NaN
        }
      } else {  // normal value
        const ftype log_f = d_REL_log2approxf(abs_orig_f);
        const ftype scaled = log_f * inv_log2eb;
        itype bin = (itype)roundf(scaled);
        const ftype abs_recon_f = d_REL_pow2approxf(bin * log2eb);
        const ftype lower = abs_orig_f / (1 + errorbound);
        const ftype upper = abs_orig_f * (1 + errorbound);
        if (!((bin >= maxbin) || (bin <= -maxbin) || (abs_orig_f >= threshold) || (abs_recon_f < lower) || (abs_recon_f > upper) || (abs_recon_f == 0) || !isfinite(abs_recon_f))) {
          bin = (bin << 1) ^ (bin >> 31);  // TCMS encoding
          bin = (bin + 1) << 1;
          if (orig_i < 0) bin |= 1;  // include sign
          output = signexpomask | bin;  // 'sign' and 'exponent' fields are all ones, 'mantissa' is non-zero (looks like a negative NaN)
        }
      }
    }
    data_out_i[i] = (output ^ signexpomask) - 1;
  }
}


#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ == 800)
static __global__ __launch_bounds__(TPB, 3)
#else
static __global__ __launch_bounds__(TPB, 2)
#endif
void d_encode(const byte* const __restrict__ input, const int insize, byte* const __restrict__ output, int* const __restrict__ outsize, int* const __restrict__ fullcarry, const float errorbound, const float threshold)
{
  // allocate shared memory buffer
  __shared__ long long chunk [3 * (CS / sizeof(long long))];

  // split into 3 shared memory buffers
  byte* in = (byte*)&chunk[0 * (CS / sizeof(long long))];
  byte* out = (byte*)&chunk[1 * (CS / sizeof(long long))];
  byte* const temp = (byte*)&chunk[2 * (CS / sizeof(long long))];

  // initialize
  const int tid = threadIdx.x;
  const int last = 3 * (CS / sizeof(long long)) - 2 - WS;
  const int chunks = (insize + CS - 1) / CS;  // round up
  long long* const head_out = (long long*)output;
  unsigned short* const size_out = (unsigned short*)&head_out[2];
  byte* const data_out = (byte*)&size_out[chunks];

  // loop over chunks
  do {
    // assign work dynamically
    if (tid == 0) chunk[last] = atomicAdd(&g_chunk_counter, 1);
    __syncthreads();  // chunk[last] produced, chunk consumed

    // terminate if done
    const int chunkID = chunk[last];
    const int base = chunkID * CS;
    if (base >= insize) break;

    // load chunk
    const int osize = min(CS, insize - base);
    long long* const input_l = (long long*)&input[base];
    long long* const out_l = (long long*)out;
    for (int i = tid; i < osize / 8; i += TPB) {
      out_l[i] = input_l[i];
    }
    const int extra = osize % 8;
    if (tid < extra) out[osize - extra + tid] = input[base + osize - extra + tid];

    // encode chunk
    __syncthreads();  // chunk produced, chunk[last] consumed
    int csize = osize;
    bool good = true;
    if (good) {
      byte* tmp = in; in = out; out = tmp;
      d_QREL_4(csize, in, out, errorbound, threshold);
      __syncthreads();
    }
    if (good) {
      byte* tmp = in; in = out; out = tmp;
      d_DIFFNB_4(csize, in, out, temp);
      __syncthreads();
    }
    if (good) {
      byte* tmp = in; in = out; out = tmp;
      d_BIT_4(csize, in, out, temp);
      __syncthreads();
    }
    if (good) {
      byte* tmp = in; in = out; out = tmp;
      good = d_RZE_1(csize, in, out, temp);
      __syncthreads();
    }

    // handle carry
    if (!good || (csize >= osize)) csize = osize;
    propagate_carry(csize, chunkID, fullcarry, (int*)temp);

    // reload chunk if incompressible
    if (tid == 0) size_out[chunkID] = csize;
    if (csize == osize) {
      // store original data
      long long* const out_l = (long long*)out;
      for (int i = tid; i < osize / 8; i += TPB) {
        out_l[i] = input_l[i];
      }
      const int extra = osize % 8;
      if (tid < extra) out[osize - extra + tid] = input[base + osize - extra + tid];
    }
    __syncthreads();  // "out" done, temp produced

    // store chunk
    const int offs = (chunkID == 0) ? 0 : *((int*)temp);
    s2g(&data_out[offs], out, csize);

    // finalize if last chunk
    if ((tid == 0) && (base + CS >= insize)) {
      // output header
      head_out[0] = (long long)insize;
      float* const head_out_f = (float*)&head_out[1];
      head_out_f[0] = errorbound;
      // compute compressed size
      *outsize = &data_out[fullcarry[chunkID]] - output;
    }
  } while (true);
}


struct GPUTimer
{
  hipEvent_t beg, end;
  GPUTimer() {hipEventCreate(&beg); hipEventCreate(&end);}
  ~GPUTimer() {hipEventDestroy(beg); hipEventDestroy(end);}
  void start() {hipEventRecord(beg, 0);}
  double stop() {hipEventRecord(end, 0); hipEventSynchronize(end); float ms; hipEventElapsedTime(&ms, beg, end); return 0.001 * ms;}
};


static void CheckCuda(const int line)
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d on line %d: %s\n\n", e, line, hipGetErrorString(e));
    throw std::runtime_error("LC error");
  }
}


int main(int argc, char* argv [])
{
  printf("PFPL GPU Single-Precision REL Compressor\n");
  printf("Copyright 2025 Texas State University\n\n");

  // read input from file
  if (argc < 4 || argc > 5) {printf("USAGE: %s input_file_name compressed_file_name error_bound [threshold]\n\n", argv[0]);  throw std::runtime_error("LC error");}

  FILE* const fin = fopen(argv[1], "rb");
  fseek(fin, 0, SEEK_END);
  const long long fsize = ftell(fin);
  if (fsize <= 0) {fprintf(stderr, "ERROR: input file too small\n\n"); throw std::runtime_error("LC error");}
  if (fsize >= 2147221529) {fprintf(stderr, "ERROR: input file too large\n\n"); throw std::runtime_error("LC error");}
  byte* const input = new byte [fsize];
  fseek(fin, 0, SEEK_SET);
  const int insize = fread(input, 1, fsize, fin);  assert(insize == fsize);
  fclose(fin);
  printf("original size: %d bytes\n", insize);

  // get GPU info
  hipSetDevice(0);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {fprintf(stderr, "ERROR: no CUDA capable device detected\n\n"); throw std::runtime_error("LC error");}
  const int SMs = deviceProp.multiProcessorCount;
  const int mTpSM = deviceProp.maxThreadsPerMultiProcessor;
  const int blocks = SMs * (mTpSM / TPB);
  const int chunks = (insize + CS - 1) / CS;  // round up
  CheckCuda(__LINE__);
  const int maxsize = 3 * sizeof(int) + chunks * sizeof(short) + chunks * CS;

  // allocate GPU memory
  byte* dencoded;
  hipHostMalloc((void **)&dencoded, maxsize);
  byte* d_input;
  hipMalloc((void **)&d_input, insize);
  hipMemcpy(d_input, input, insize, hipMemcpyHostToDevice);
  byte* d_encoded;
  hipMalloc((void **)&d_encoded, maxsize);
  int* d_encsize;
  hipMalloc((void **)&d_encsize, sizeof(int));
  int* d_fullcarry;
  hipMalloc((void**)&d_fullcarry, chunks * sizeof(int));
  CheckCuda(__LINE__);

  const float errorbound = atof(argv[3]);
  const float threshold = (argc == 5) ? atof(argv[4]) : std::numeric_limits<float>::infinity();
  if (threshold < std::numeric_limits<float>::min()) {printf("ERROR: threshold must be a positive, normal, floating-point value\n");  throw std::runtime_error("LC error");}
  GPUTimer dtimer;
  for (int i = 0; i < NUM_RUNS; i++) {
    hipDeviceSynchronize();
    dtimer.start();

    d_reset<<<1, 1>>>();
    hipMemset(d_fullcarry, 0, chunks * sizeof(byte));
    d_encode<<<blocks, TPB>>>(d_input, insize, d_encoded, d_encsize, d_fullcarry, errorbound, threshold);

    hipDeviceSynchronize();
    double runtime = dtimer.stop();

    if (i < NUM_RUNS - 1) {
      hipMemset(d_encsize, 0, sizeof(int));
      hipMemset(d_encoded, 0, maxsize);
    }
    CheckCuda(__LINE__);
    printf("lc comp ecltime, %12.9f\n", runtime);
  }

  // get encoded GPU result
  int dencsize = 0;
  hipMemcpy(&dencsize, d_encsize, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(dencoded, d_encoded, dencsize, hipMemcpyDeviceToHost);
  printf("encoded size: %d bytes\n", dencsize);
  printf("compression ratio: %.2f\n", 1.0 * insize / dencsize);
  CheckCuda(__LINE__);

  // write to file
  FILE* const fout = fopen(argv[2], "wb");
  fwrite(dencoded, 1, dencsize, fout);
  fclose(fout);

  // clean up GPU memory
  hipFree(d_input);
  hipFree(d_encoded);
  hipFree(d_encsize);
  CheckCuda(__LINE__);

  // clean up
  delete [] input;
  hipHostFree(dencoded);
  return 0;
}
